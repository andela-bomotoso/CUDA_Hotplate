/* A program to solve the hotplate problem  using GPU

  Author: Bukola Grace Omotoso
  MNumber: M01424979
  ID: bgo2e
  Last Modified: 11/27/2018
  
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>
#include<hip/hip_runtime.h>





#define CHECK(call) \
{ \
 const hipError_t error = call; \
 if (error != hipSuccess) \
 { \
 printf("Error: %s:%d, ", __FILE__, __LINE__); \
 printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
 exit(1); \
 } \
}


float** buildHotplate(int rows, int columns) {
float** hotplate;
hotplate = (float**) malloc(rows*sizeof(float*));
for (int i = 0; i < rows; i++)
   hotplate[i] = (float*) malloc(columns*sizeof(float));
   return hotplate;
}

float* flattenArray(float** arrayToFlatten, int num_rows, int num_cols){
	float* flattenedArray = (float*) malloc(num_rows*num_cols*sizeof(float));
	int counter = 0;
	
	for (int row = 0; row < num_rows; row++){
		
	{
		for(int col = 0; col < num_cols; col++){
			flattenedArray[counter] = arrayToFlatten[row][col];
			counter++;
		}
	}
	
}
		return flattenedArray;
}

 void initializeHotPlate(int num_rows, int num_cols, float** hotplate, float** hotplateClone, int top_temp, int left_temp, int right_temp, int bottom_temp)	{
 	int num_outer_grid = (2 * num_rows) + (2 * (num_cols - 2));
	float outer_grid_sum = (top_temp * (num_cols - 2)) + (left_temp * (num_rows - 1)) + (bottom_temp * num_cols) + (right_temp * (num_rows - 1));
    float initial_inner_val = outer_grid_sum / num_outer_grid;

    for (int row = 0; row < num_rows; row++) {
            for (int column = 0; column < num_cols; column++) {

                //top values override the top row except the edges
                 if ((row == 0) & (column != 0 & column != num_cols - 1)) {
                	hotplate[row][column] = top_temp;
                    hotplateClone[row][column] = top_temp;
                } 
                else if (column == 0 && (row != (num_rows-1))) {
                    hotplate[row][column] = left_temp;
                    hotplateClone[row][column] = left_temp;
                }
                else if (column == (num_cols - 1) && (row != (num_rows-1))) {
                    hotplate[row][column] = right_temp;
                    hotplateClone[row][column] = right_temp;
                }
                else if(row == (num_rows -1 )){
                    hotplate[row][column] = bottom_temp;
                    hotplateClone[row][column] = bottom_temp;
                }
                if ((row != 0) && (row != num_rows - 1) && (column != 0) && (column != num_cols - 1))
                    hotplate[row][column] = initial_inner_val;
            }
        }

 }

 void swapHotplate(float *a, float *b) {

    float *tmp = a;
    a = b;
    b = tmp;
}


__global__ 
void generateHeat(int num_rows, int num_cols, float* hotplate, float* hotplateClone, float* d_maximums, float epsilon) {
        float max_difference = 0;
        float previous_val;
        float current_val;
        float diff;
        
         int row = blockIdx.x * blockDim.x + threadIdx.x;
         
            if (row > 0 && row < (num_rows-1)){
                for (int col = 1; col < (num_cols - 1); col++) {
					
				int idx = (row * num_cols) + col;
				float top = hotplate[idx - num_cols];				
				float bottom = hotplate[idx + num_cols];
				float left =  hotplate[idx - 1];
				float right =  hotplate[idx + 1];

                    previous_val = hotplate[idx];
                    current_val = (top + bottom + left + right) / 4.0;
                    diff = fabsf(previous_val - current_val);
                    if (diff > max_difference){
                        max_difference = diff;
                    }
                    hotplateClone[idx] = current_val;
                }
                
                d_maximums[row] = max_difference;
			}
}


/*Get the maximum values from all threads*/
float max_max_diff(float arr[], int n)
{
    int i;
        float max = arr[0];
    for (i = 1; i < n; i++)
        if (arr[i] > max)
            max = arr[i];
    return max;
}



int main(int argc, char const *argv[])
{		int num_rows = atoi(argv[1]);
		int num_cols = atoi(argv[2]);
		int top_temp = atoi(argv[3]);
		int left_temp = atoi(argv[4]);
		int right_temp = atoi(argv[5]);
		int bottom_temp = atoi(argv[6]);
		float epsilon = atof(argv[7]);
		
		float* flattenedhotplate;
		float* flattenedhotplateClone;
		float* maximums;

		
		int gridsize = 8;
		int block = 0;
		int block1 = (num_rows/gridsize);
		if (gridsize > num_rows)
			block = 1;
		else if ((block1 * gridsize) < num_rows){
			block = block1 + 1;
		}else{
			block = block1;
		}
		size_t nBytes = num_rows*num_cols * sizeof(float);
		double max_difference = epsilon + 1;
		int counter = 0;

        float** hotplate =  buildHotplate(num_rows, num_cols);
        float** hotplateClone = buildHotplate(num_rows, num_cols);
        
        
         
        initializeHotPlate(num_rows, num_cols, hotplate, hotplateClone, top_temp, left_temp, right_temp, bottom_temp);    
         
         flattenedhotplate = (float*) malloc(num_cols*num_rows*sizeof(float));
         flattenedhotplateClone = (float*) malloc(num_cols*num_rows*sizeof(float));
         maximums = (float*)malloc(num_rows*sizeof(float));
         
     
         flattenedhotplate = flattenArray(hotplate, num_rows, num_cols);
         flattenedhotplateClone = flattenArray(hotplateClone, num_rows, num_cols);

         
         float *d_hotplate; 
         float *d_hotplateClone;
         float *d_maximums;
	

		CHECK(hipMalloc((float**)&d_hotplate, nBytes));
		CHECK(hipMalloc((float**)&d_hotplateClone, nBytes));
		CHECK(hipMalloc((float**)&d_maximums, num_rows*sizeof(float)));
		
		
		CHECK(hipMemcpy( d_hotplate, flattenedhotplate, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_hotplateClone, flattenedhotplateClone, nBytes, hipMemcpyHostToDevice));
		
		printf("%10s%10s\n", "Iteration", "Epsilon");

		 while(max_difference > epsilon){
        
		generateHeat<<<block,gridsize>>>(num_rows, num_cols, d_hotplate, d_hotplateClone, d_maximums, epsilon);
	
		hipDeviceSynchronize();
		CHECK(hipMemcpy(maximums, d_maximums, num_rows*sizeof(float), hipMemcpyDeviceToHost));
		max_difference = max_max_diff(maximums, num_rows-1);
        
            float *T = d_hotplate;
            d_hotplate = d_hotplateClone;
            d_hotplateClone  =  T;
            
            if (counter > 0 && (counter & (counter - 1)) == 0)
                printf("%6d%15.6f\n", counter, max_difference);
            if (max_difference < epsilon) {
                printf("%6d%15.6f\n", counter, max_difference);
                break;
            }
            counter++;
	}
	
	hipFree(d_hotplate);
	hipFree(d_hotplateClone);
	hipFree(d_maximums);
	
		return 0;

}

